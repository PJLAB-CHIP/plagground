#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"
#include "playground/utils.hpp"

namespace playground
{
PG_MATMUL_SIG(float16_t, CUBLAS_VER, M, N, K, A, B, C)
{
    const float16_t Alpha = 1.0f;
    const float16_t Beta = 0.0f;
    hipblasGemmEx(s_getCublasHandle<float16_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
                 K, &Alpha, B, HIP_R_16F, N, A, HIP_R_16F, K, &Beta, C,
                 HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground
