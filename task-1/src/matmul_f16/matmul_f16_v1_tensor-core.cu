#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"
#include "playground/utils.hpp"

namespace playground
{
template <>
void matmul<float16_t, 1>(const size_t M, const size_t N, const size_t K,
                          const float16_t* const A, const float16_t* const B,
                          float16_t* const C)
{
    const float16_t Alpha = 1.0f;
    const float16_t Beta = 0.0f;
    hipblasGemmEx(s_getCublasHandle<float16_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
                 K, &Alpha, B, HIP_R_16F, N, A, HIP_R_16F, K, &Beta, C,
                 HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground
