#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/utils.hpp"

namespace playground
{
template <>
void matmul<float16_t, 1>(const size_t m, const size_t n, const size_t k, const float16_t* const A,
                          const float16_t* const B, float16_t* const C)
{
    const float16_t alpha = 1.0f;
    const float16_t beta = 0.0f;
    hipblasGemmEx(s_getCublasHandle<float16_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, B,
                 HIP_R_16F, n, A, HIP_R_16F, k, &beta, C, HIP_R_16F, n, HIP_R_16F,
                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground
