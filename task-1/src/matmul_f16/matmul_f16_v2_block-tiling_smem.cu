#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include "playground/common.hpp"
#include "playground/hyperparams.hpp"
#include "playground/matmul.hpp"
#include "playground/ptx.hpp"
#include "playground/system.hpp"
#include "playground/utils.hpp"

namespace playground
{
#ifdef USE_WMMA
static constexpr size_t FragmentM = 16;
static constexpr size_t FragmentN = 16;
static constexpr size_t FragmentK = 16;
#else
static constexpr size_t FragmentM = 16;
static constexpr size_t FragmentN = 8;
static constexpr size_t FragmentK = 16;
#endif

static constexpr size_t WarpSize = device::WRAP_SIZE;

__global__ void hgemmV2_kernel(const float16_t* const A, const float16_t* const B,
                               float16_t* const C, const size_t M, const size_t N, const size_t K)
{
#ifdef USE_WMMA
    using namespace nvcuda;

    const size_t K_tiles = ceilDivide(K, FRAGMENT_K);

    const size_t warp_row = blockIdx.y * FRAGMENT_M;
    const size_t warp_col = blockIdx.x * FRAGMENT_N;

    if (warp_row >= M && warp_col >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, FRAGMENT_M, FRAGMENT_N, FRAGMENT_K, float16_t> C_frag;

    wmma::fill_fragment(C_frag, 0.0);

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        wmma::fragment<wmma::matrix_a, FRAGMENT_M, FRAGMENT_N, FRAGMENT_K, float16_t,
                       wmma::row_major>
            A_frag;
        wmma::fragment<wmma::matrix_b, FRAGMENT_M, FRAGMENT_N, FRAGMENT_K, float16_t,
                       wmma::col_major>
            B_frag;

        wmma::load_matrix_sync(A_frag, A + warp_row * K + i * FRAGMENT_K, K);
        wmma::load_matrix_sync(B_frag, B + i * FRAGMENT_K + warp_col * K, K);

        wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
    }

    wmma::store_matrix_sync(C + warp_row * N + warp_col, C_frag, N, wmma::mem_row_major);

#else
    const size_t K_tiles = ceilDivide(K, FragmentK);

    const size_t warp_row = blockIdx.y * FragmentM;
    const size_t warp_col = blockIdx.x * FragmentN;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    __shared__ float16_t A_smem[FragmentM][FragmentK];
    __shared__ float16_t B_smem[FragmentN][FragmentK];
    __shared__ float16_t C_smem[FragmentM][FragmentN];

    static_assert((FRAGMENT_M * FRAGMENT_K + FRAGMENT_N * FRAGMENT_K + FRAGMENT_M * FRAGMENT_N) *
                          sizeof(float16_t) <=
                      device::SM_PER_BLOCK,
                  "Not enough shared memory");

    const size_t lane_id = threadIdx.x % WarpSize;

    uint32_t RC[2] = {0, 0};

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        // Load A tiles to shared memory
        *(rCast<int4*>(&A_smem[lane_id / 2][0]) + lane_id % 2) =
            *(rCast<const int4*>(&A[(warp_row + lane_id / 2) * K + i * FragmentK]) + lane_id % 2);
        // Load B tiles to shared memory
        if (lane_id < FragmentN * 2) {
            *(rCast<int4*>(&B_smem[lane_id / 2][0]) + lane_id % 2) =
                *(rCast<const int4*>(&B[i * FragmentK + (warp_col + lane_id / 2) * K]) +
                  lane_id % 2);
        }
        __syncthreads();

        uint32_t RA[4];  // 4*32b -> 8 float16_t
        uint32_t RB[2];  // 2*32b -> 4 float16_t

        uint32_t A_smem_lane_addr =
            __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_smem_lane_addr);

        uint32_t B_smem_lane_addr =
            __cvta_generic_to_shared(&B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
        LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }

    *(rCast<uint32_t*>(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
    *(rCast<uint32_t*>(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

    __syncthreads();

    if (lane_id < FragmentM) {
        rCast<int4>(C[(warp_row + lane_id) * N + warp_col]) = rCast<int4>(C_smem[lane_id][0]);
    }
#endif
}

template <>
void matmul<float16_t, 2>(const size_t m, const size_t n, const size_t k, const float16_t* const A,
                          const float16_t* const B, float16_t* const C)
{
    dim3 blockSize(WarpSize);
    dim3 gridSize(ceilDivide(n, FragmentN), ceilDivide(m, FragmentM));
    hgemmV2_kernel<<<gridSize, blockSize>>>(A, B, C, m, n, k);
}
}  // namespace playground