#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_SIG(float32_t, 1, M, N, K, A, B, C)
{
    const float32_t Alpha = 1.0f;
    const float32_t Beta = 0.0f;
    hipblasSgemm(s_getCublasHandle<float32_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
                K, &Alpha, B, N, A, K, &Beta, C, N);
}
}  // namespace playground