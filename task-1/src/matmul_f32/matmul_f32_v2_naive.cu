#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_vector_types.h>

#include "playground/matmul.hpp"
#include "playground/system.hpp"
#include "playground/utils/address.hpp"

namespace playground
{

__global__ void sgemmV2_kernel(const float32_t* const A,
                               const float32_t* const B, float32_t* const C,
                               const size_t M, const size_t N, const size_t K)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        float psum = 0.0;
#pragma unroll
        for (int k = 0; k < K; k++) {
            psum += A[computeOffset<uint32_t>(m, k, M, K)] *
                    B[computeOffset<uint32_t>(k, n, K, N)];
        }
        C[computeOffset<uint32_t>(m, n, M, N)] = psum;
    }
}

template <>
void matmul<float32_t, 2>(const size_t M, const size_t N, const size_t K,
                          const float32_t* const A, const float32_t* const B,
                          float32_t* const C)
{
    const dim3 BlockSize(32, 32);
    const dim3 GridSize(128, 128);
    sgemmV2_kernel<<<GridSize, BlockSize>>>(A, B, C, M, N, K);
}
}  // namespace playground